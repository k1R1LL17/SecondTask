#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cstdlib>
#include <chrono>
#include <hip/hip_runtime.h>

__global__ void multiplyMatrices(int* firstMatrix, int* secondMatrix, int* resultMatrix, int sizeOfMatrix) {
    int elementIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (elementIndex < sizeOfMatrix) {
        resultMatrix[elementIndex] = firstMatrix[elementIndex] * secondMatrix[elementIndex];
    }
}

int main() {
    int matrixSize = 4096;
    int memorySize = matrixSize * sizeof(int);

    int* hostMatrixA = (int*)malloc(memorySize);
    int* hostMatrixB = (int*)malloc(memorySize);
    int* hostResultMatrix = (int*)malloc(memorySize);

    srand(time(0));

    for (int elementIndex = 0; elementIndex < matrixSize; elementIndex++) {
        hostMatrixA[elementIndex] = rand() % 100 + 1;
        hostMatrixB[elementIndex] = rand() % 100 + 1;
    }

    int* deviceMatrixA, * deviceMatrixB, * deviceResultMatrix;
    hipMalloc((void**)&deviceMatrixA, memorySize);
    hipMalloc((void**)&deviceMatrixB, memorySize);
    hipMalloc((void**)&deviceResultMatrix, memorySize);

    hipMemcpy(deviceMatrixA, hostMatrixA, memorySize, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, hostMatrixB, memorySize, hipMemcpyHostToDevice);


    auto start = std::chrono::high_resolution_clock::now();

    int threadsPerBlock = 256;
    int blocksPerGrid = (matrixSize + threadsPerBlock - 1) / threadsPerBlock;
    multiplyMatrices << <blocksPerGrid, threadsPerBlock >> > (deviceMatrixA, deviceMatrixB, deviceResultMatrix, matrixSize);

    hipMemcpy(hostResultMatrix, deviceResultMatrix, memorySize, hipMemcpyDeviceToHost);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "GPU Time: " << duration.count() << " seconds" << std::endl;

    std::cout << "First 10 elements of the result matrix:" << std::endl;
    for (int elementIndex = 0; elementIndex < 10; elementIndex++) {
        std::cout << hostResultMatrix[elementIndex] << " ";
    }
    std::cout << std::endl;

    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceResultMatrix);
    free(hostMatrixA);
    free(hostMatrixB);
    free(hostResultMatrix);

    return 0;
}
